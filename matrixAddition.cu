
#include <hip/hip_runtime.h>
#include <cstdio>

#if defined(NDEBUG)
#define CUDA_CHECK(x)	(x)
#else
#define CUDA_CHECK(x)	do {\
		(x); \
		hipError_t e = hipGetLastError(); \
		if (hipSuccess != e) { \
			printf("cuda failure \"%s\" at %s:%d\n", \
			       hipGetErrorString(e), \
			       __FILE__, __LINE__); \
			exit(1); \
		} \
	} while (0)
#endif


// kernel program for the device (GPU): compiled by NVCC
__global__ void addKernel(int* c, const int* a, const int* b) {
	int x = threadIdx.x;
	int y = threadIdx.y;
	int i = y * (blockDim.x) + x; // [y][x] = y * WIDTH + x;
	c[i] = a[i] + b[i];
}


// main program for the CPU: compiled by MS-VC++
int main(void) {
	// host-side data
	const int WIDTH = 5;
	int a[WIDTH][WIDTH];
	int b[WIDTH][WIDTH];
	int c[WIDTH][WIDTH] = { 0 };
	// make a, b matrices
	for (int y = 0; y < WIDTH; ++y) {
		for (int x = 0; x < WIDTH; ++x) {
			a[y][x] = y * 10 + x;
			b[y][x] = (y * 10 + x) * 100;
		}
	}
	// device-side data
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	// allocate device memory
	CUDA_CHECK( hipMalloc((void**)&dev_a, WIDTH * WIDTH * sizeof(int)) );
	CUDA_CHECK( hipMalloc((void**)&dev_b, WIDTH * WIDTH * sizeof(int)) );
	CUDA_CHECK( hipMalloc((void**)&dev_c, WIDTH * WIDTH * sizeof(int)) );
	// copy from host to device
	CUDA_CHECK( hipMemcpy(dev_a, a, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice) );
	CUDA_CHECK( hipMemcpy(dev_b, b, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice) );
	// launch a kernel on the GPU with one thread for each element.
	dim3 dimBlock(WIDTH, WIDTH, 1); // x, y, z
	addKernel <<< 1, dimBlock>>>(dev_c, dev_a, dev_b);		// dev_c = dev_a + dev_b;
	CUDA_CHECK( hipPeekAtLastError() );
	// copy from device to host
	CUDA_CHECK( hipMemcpy(c, dev_c, WIDTH * WIDTH * sizeof(int), hipMemcpyDeviceToHost) );
	// free device memory
	CUDA_CHECK( hipFree(dev_c) );
	CUDA_CHECK( hipFree(dev_a) );
	CUDA_CHECK( hipFree(dev_b) );
	// print the result
	for (int y = 0; y < WIDTH; ++y) {
		for (int x = 0; x < WIDTH; ++x) {
			printf("%5d", c[y][x]);
		}
		printf("\n");
	}
	// done
	return 0;
}
